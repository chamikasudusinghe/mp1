#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define NUM_RUNS 10
#define BLOCK_SIZE 16
#define TILE_SIZE 32
#define TILE_SIZE_BEST 16

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
  	int col = blockIdx.x * blockDim.x + threadIdx.x;
	// Keep the threads within the bounds of the matrix
    if (row < M && col < N) {
        float sum = 0.0f;
		// Check if the number of columns in A is equal to the number of rows in B
        for (int k = 0; k < K; k++) {
			// Multiply the row of A and the column of B
            sum += A[row * K + k] * B[k * N + col];
        }
		// Store the result in the output matrix
        C[row * N + col] = sum;
    }
}

void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {
	// Declare the shared memory
	__shared__ float shared_A[TILE_SIZE][TILE_SIZE];
	__shared__ float shared_B[TILE_SIZE][TILE_SIZE];

	int tx = threadIdx.x;
    int ty = threadIdx.y;

	// Calculate the row and column of the element in C for tiling
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

	// Initialize the value of the element in C
    float Cvalue = 0.0f;

	// Loop over the tiles of the input in A and B to compute the element in C
    for (int m = 0; m < (K + TILE_SIZE - 1) / TILE_SIZE; m++) {
		// Load the tiles of the input in A and B to the shared memory
        if (row < M && m * TILE_SIZE + tx < K) {
            shared_A[ty][tx] = A[row * K + m * TILE_SIZE + tx];
        } else {
            shared_A[ty][tx] = 0.0f; 
        }
		// Load the tiles of the input in B to the shared memory
        if (col < N && m * TILE_SIZE + ty < K) {
            shared_B[ty][tx] = B[(m * TILE_SIZE + ty) * N + col];
        } else {
            shared_B[ty][tx] = 0.0f; 
        }
		// Synchronize the threads
        __syncthreads();
		// Compute the value of the element in C
        for (int k = 0; k < TILE_SIZE; k++) {
            Cvalue += shared_A[ty][k] * shared_B[k][tx];
        }
		// Synchronize the threads
        __syncthreads();
    }
	// Store the value of the element in C if it is within the boundary
    if (row < M && col < N) {
        C[row * N + col] = Cvalue;
    }
}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
	// Declare the shared memory
	__shared__ float shared_A[TILE_SIZE_BEST][TILE_SIZE_BEST];
	__shared__ float shared_B[TILE_SIZE_BEST][TILE_SIZE_BEST];

	int tx = threadIdx.x;
    int ty = threadIdx.y;

	// Calculate the row and column of the element in C
    int row = blockIdx.y * TILE_SIZE_BEST + ty;
    int col = blockIdx.x * TILE_SIZE_BEST + tx;

	// Initialize the value of the element in C
    float Cvalue = 0.0f;

	// Loop over the tiles of the input in A and B to compute the element in C
    for (int m = 0; m < (K + TILE_SIZE_BEST - 1) / TILE_SIZE_BEST; m++) {
		// Load the tiles of the input in A and B to the shared memory
        if (row < M && m * TILE_SIZE_BEST + tx < K) {
            shared_A[ty][tx] = A[row * K + m * TILE_SIZE_BEST + tx];
        } else {
            shared_A[ty][tx] = 0.0f; 
        }
		// Load the tiles of the input in B to the shared memory
        if (col < N && m * TILE_SIZE_BEST + ty < K) {
            shared_B[ty][tx] = B[(m * TILE_SIZE_BEST + ty) * N + col];
        } else {
            shared_B[ty][tx] = 0.0f; 
        }
		// Synchronize the threads
        __syncthreads();
		// Compute the value of the element in C
        for (int k = 0; k < TILE_SIZE_BEST; k++) {
            Cvalue += shared_A[ty][k] * shared_B[k][tx];
        }
		// Synchronize the threads
        __syncthreads();
    }
	// Store the value of the element in C if it is within the boundary
    if (row < M && col < N) {
        C[row * N + col] = Cvalue;
    }
}

void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(TILE_SIZE_BEST, TILE_SIZE_BEST);
    dim3 gridSize((N + TILE_SIZE_BEST - 1) / TILE_SIZE_BEST, (M + TILE_SIZE_BEST - 1) / TILE_SIZE_BEST);
    gemm_gpu_o3_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

void gemm_cublas(float* A, float* B, float* C, int M, int N, int K) {
	// Create a cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	// Define scalars for matrix multiplication 
	const float alpha = 1.0f;
	const float beta = 0.0f;
	// Perform matrix multiplication using cuBLAS
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
	// Destroy the cuBLAS handle
	hipblasDestroy(handle);
}


int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)
	CHECK(gemm_cublas)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)
	TIME(gemm_cublas)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}